#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utils.h"

void read_image(char* path, float** image, int image_nums) {
    int size;
    FILE* fp = fopen(path, "rb");
    // skip_header(fp, IMAGE_HEADER_LENGTH);
    int length;
    unsigned int* header = (unsigned int*) malloc(IMAGE_HEADER_LENGTH * sizeof(unsigned int));
    length = fread(header, sizeof(unsigned int), IMAGE_HEADER_LENGTH, fp);
    assert(length == IMAGE_HEADER_LENGTH);
    free(header);

    unsigned char* data = (unsigned char*) malloc(IMAGE_SIZE * sizeof(unsigned char));
    for (int i=0; i<image_nums; i++) {
        size = fread(data, sizeof(unsigned char), IMAGE_SIZE, fp);
        assert(size == IMAGE_SIZE);
        for (int j=0; j<IMAGE_SIZE; j++) {
            assert(data[j] <= MAX_BRIGHTNESS);
            image[i][j] = (float)data[j] / MAX_BRIGHTNESS;
        }
    }
    free(data);
    fclose(fp);
}

void read_label(char* path, int* label, int label_nums) {
    int size;
    FILE* fp = fopen(path, "rb");
    // skip_header(fp, LABEL_HEADER_LENGTH);
    int length;
    unsigned int* header = (unsigned int*) malloc(LABEL_HEADER_LENGTH * sizeof(unsigned int));
    length = fread(header, sizeof(unsigned int), LABEL_HEADER_LENGTH, fp);
    assert(length == LABEL_HEADER_LENGTH);
    free(header);

    unsigned char* data = (unsigned char*) malloc(label_nums * LABEL_SIZE * sizeof(unsigned char));
    size = fread(data, sizeof(unsigned char), label_nums, fp);
    assert(size == label_nums);
    for (int i=0; i<label_nums; i++) {
        assert(data[i] <= 9);
        label[i] = (int)data[i];
    }
    free(data);
    fclose(fp);
}

void read_data_set(float** data_set, char* data_path, int* labels, char* label_path, int data_set_size) {
    for (int i = 0; i < data_set_size; i++) {
        assert(hipHostMalloc((void**) &data_set[i], sizeof(float) * IMAGE_SIZE) == hipSuccess);
    }
    read_image(data_path, data_set, data_set_size);
    read_label(label_path, labels, data_set_size);
}

void free_data_set(float** data_set, int* labels, int data_set_size) {
    for (int i = 0; i < data_set_size; i++) {
        assert(hipHostFree(data_set[i]) == hipSuccess);
    }
    assert(hipHostFree(data_set) == hipSuccess);
    assert(hipHostFree(labels) == hipSuccess);
}

float gaussian(float mu, float sigma) {
    const float epsilon = 1e-10;

    static float z1;
    static int generate = 0;
    generate = !generate;

    if (!generate)
       return z1 * sigma + mu;

    float u1, u2;
    do {
       u1 = rand() * (1.0 / RAND_MAX);
       u2 = rand() * (1.0 / RAND_MAX);
    } while (u1 <= epsilon);

    float z0;
    z0 = sqrt(-2.0 * log(u1)) * cos(2.0 * M_PI * u2);
    z1 = sqrt(-2.0 * log(u1)) * sin(2.0 * M_PI * u2);
    return z0 * sigma + mu;
}

void kaiming_init(float *data, int m, int n) {
    float std = sqrt(2.0 / n);
    for(int i = 0; i < m * n; i++) {
        data[i] = gaussian(0, std);
    }
}

__device__ float get_val(float* vec, int row, int col, int n) {
    return vec[row * n + col];
}

__device__ void set_val(float* vec, int row, int col, int n, float value) {
    vec[row * n + col] = value;
}

__device__ void add_val(float* vec, int row, int col, int n, float value) {
    vec[row * n + col] += value;
}

int argmax(float* data, int length) {
    int max_idx = 0;
    float max_val = data[0];
    for (int i=1; i<length; i++) {
        if (data[i] > max_val) {
            max_idx = i;
            max_val = data[i];
        }
    }
    return max_idx;
}

float cross_entropy_loss(int y, float* y_hat) {
    return -log(y_hat[y]);
}

__global__ void fc_layer(float* in, float* out, float* w, int w_width, int w_height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < w_height && j < w_width) {
        float value = get_val(w, i, j, w_width);
        atomicAdd(&out[i], in[j] * value);
    }
}

__global__ void relu(float* in, int in_dimension, float* out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < in_dimension) {
        out[i] = in[i] > 0 ? in[i] : 0;
    }
}

void softmax(float* in, int in_dimension, float* out, int out_dimension) {
    float sum = 0;
    float max_val = in[0];
    for (int i=1; i<in_dimension; i++) {
        if (in[i] > max_val) {
            max_val = in[i];
        }
    }
    for (int i=0; i<in_dimension; i++) {
        out[i] = exp(in[i] - max_val);
        sum += out[i];
    }
    for (int i=0; i<in_dimension; i++) {
        out[i] /= sum;
    }
}

__global__ void out_backprop(float* out, float* grad_out, int out_size, int label) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < out_size) {
        grad_out[i] = out[i] - (i==label);
    }
}

__global__ void neuron_backprop(float* in, float* grad_out, float* grad_in, float* w, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < height && j < width) {
        float value = get_val(w, i, j, width);
        atomicAdd(&grad_in[j], value * grad_out[i]);
    }
}

__global__ void relu_backprop(float* in, float* grad_in, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        grad_in[i] = in[i] > 0 ? grad_in[i] : 0;
    }
}

__global__ void weight_backprop(float* in, float* grad_out, float* grad_w, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < height && j < width) {
        add_val(grad_w, i, j, width, in[j] * grad_out[i]);
    }
}

__global__ void bias_backprop(float* grad_out, float* grad_bias, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        grad_bias[i] += grad_out[i];
    }
}

void allocate_weight_and_data(float** weight_cu, float** bias_cu, float** data_cu, int hidden_size, int num_layers, int data_size) {
    assert(hipMalloc((void**)weight_cu, IN_DIMENSION * hidden_size * sizeof(float)) == hipSuccess);
    assert(hipMalloc((void**)bias_cu, hidden_size * sizeof(float)) == hipSuccess);
    for (int i = 1; i < num_layers-1; ++i) {
        assert(hipMalloc((void**)(&weight_cu[i]), hidden_size * hidden_size * sizeof(float)) == hipSuccess);
        assert(hipMalloc((void**)(&bias_cu[i]), hidden_size * sizeof(float)) == hipSuccess);
    }
    assert(hipMalloc((void**)(&weight_cu[num_layers-2]), hidden_size * OUT_DIMENSION * sizeof(float)) == hipSuccess);
    assert(hipMalloc((void**)(&bias_cu[num_layers-2]), OUT_DIMENSION * sizeof(float)) == hipSuccess);

    for (int i = 0; i < data_size; ++i) {
        assert(hipMalloc((void**)(&data_cu[i]), IMAGE_SIZE * sizeof(float)) == hipSuccess);
    }
}

void allocate_memory(float** neurons, float** grad_weight, float** grad_bias, float** grad_neuron,
                        float** weight_cu, float** bias_cu, float** data_cu, int hidden_size, int num_layers, int data_size) {
    assert(hipMalloc((void**)neurons, IN_DIMENSION * sizeof(float)) == hipSuccess);
    assert(hipMalloc((void**)grad_neuron, IN_DIMENSION * sizeof(float)) == hipSuccess);
    assert(hipMalloc((void**)grad_weight, IN_DIMENSION * hidden_size * sizeof(float)) == hipSuccess);
    assert(hipMalloc((void**)grad_bias, hidden_size * sizeof(float)) == hipSuccess);

    for (int i = 1; i < num_layers-1; ++i) {
        assert(hipMalloc((void**)(&neurons[i]), hidden_size * sizeof(float)) == hipSuccess);
        assert(hipMalloc((void**)(&grad_neuron[i]), hidden_size * sizeof(float)) == hipSuccess);
        if (i < num_layers-2) {
            assert(hipMalloc((void**)(&grad_weight[i]), hidden_size * hidden_size * sizeof(float)) == hipSuccess);
            assert(hipMalloc((void**)(&grad_bias[i]), hidden_size * sizeof(float)) == hipSuccess);
        }
    }

    assert(hipMalloc((void**)(&neurons[num_layers-1]), OUT_DIMENSION * sizeof(float)) == hipSuccess);
    assert(hipMalloc((void**)(&grad_neuron[num_layers-1]), OUT_DIMENSION * sizeof(float)) == hipSuccess);
    assert(hipMalloc((void**)(&grad_weight[num_layers-2]), hidden_size * OUT_DIMENSION * sizeof(float)) == hipSuccess);
    assert(hipMalloc((void**)(&grad_bias[num_layers-2]), OUT_DIMENSION * sizeof(float)) == hipSuccess);

    allocate_weight_and_data(weight_cu, bias_cu, data_cu, hidden_size, num_layers, data_size);
}

void free_weight_and_data(float** weight_cu, float** bias_cu, float** data_cu, int num_layers, int data_size) {
    for (int i = 0; i < num_layers-1; ++i) {
        assert(hipFree(weight_cu[i]) == hipSuccess);
        assert(hipFree(bias_cu[i]) == hipSuccess);
    }
    for (int i = 0; i < data_size; ++i) {
        assert(hipFree(data_cu[i]) == hipSuccess);
    }

    assert(hipHostFree(weight_cu) == hipSuccess);
    assert(hipHostFree(bias_cu) == hipSuccess);
    assert(hipHostFree(data_cu) == hipSuccess);
}

void free_memory(float** neurons, float** grad_weight, float** grad_bias, float** grad_neuron, float** weight_cu, float** bias_cu, float** data_cu, int num_layers, int data_size) {
    for (int i = 0; i < num_layers; ++i) {
        assert(hipFree(neurons[i]) == hipSuccess);
        assert(hipFree(grad_neuron[i]) == hipSuccess);
        if (i < num_layers-1) {
            assert(hipFree(grad_weight[i]) == hipSuccess);
            assert(hipFree(grad_bias[i]) == hipSuccess);
        }
    }

    assert(hipHostFree(neurons) == hipSuccess);
    assert(hipHostFree(grad_weight) == hipSuccess);
    assert(hipHostFree(grad_bias) == hipSuccess);
    assert(hipHostFree(grad_neuron) == hipSuccess);

    free_weight_and_data(weight_cu, bias_cu, data_cu, num_layers, data_size);
}