#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdio>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "utils.h"

char TRAIN_IMAGE_FILE[] = "../data/train-images-idx3-ubyte";
char TRAIN_LABEL_FILE[] = "../data/train-labels-idx1-ubyte";
char TEST_IMAGE_FILE[] = "../data/t10k-images-idx3-ubyte";
char TEST_LABEL_FILE[] = "../data/t10k-labels-idx1-ubyte";

int forward(float* image, int image_size, int label, float** weight, float** bias,
            float** neurons, int hidden_size, int num_layers, float* loss, int thread_num) {
    assert(hipMemcpy(neurons[0], image, image_size * sizeof(float), hipMemcpyDeviceToDevice) == hipSuccess);
    for (int i = 0; i < num_layers-1; i++) {
        int width = (i == 0 ? image_size : hidden_size);
        int height = (i == num_layers-2 ? OUT_DIMENSION : hidden_size);

        int block_num = ceil(sqrt((float)thread_num));
        dim3 dim_block(block_num, block_num);
        dim3 dim_grid(ceil((float)height/block_num), ceil((float)width/block_num));
        int thread_num_1d = (height > thread_num ? thread_num : height);
        int block_num_1d = ceil((float)height / thread_num_1d);

        assert(hipMemcpy(neurons[i+1], bias[i], height * sizeof(float), hipMemcpyDeviceToDevice) == hipSuccess);
        fc_layer<<<dim_grid, dim_block>>>(neurons[i], neurons[i+1], weight[i], width, height);

        if (i < num_layers-2) {
            relu<<<block_num_1d, thread_num_1d>>>(neurons[i+1], height, neurons[i+1]);
        }
    }

    float out[OUT_DIMENSION];
    assert(hipMemcpy(out, neurons[num_layers-1], OUT_DIMENSION * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess);
    softmax(out, OUT_DIMENSION, out, OUT_DIMENSION);
    assert(hipMemcpy(neurons[num_layers-1], out, OUT_DIMENSION * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);

    *loss = cross_entropy_loss(label, out);

    return argmax(out, OUT_DIMENSION);
}

void backward(int y, float** weight, float** bias, float** neuron, float** grad_weight,
            float** grad_bias, float** grad_neuron, int hidden_size, int num_layers, int thread_num) {
    // output layer
    int output_layer_idx = num_layers - 1;
    int thread_num_1d = (OUT_DIMENSION > thread_num ? thread_num : OUT_DIMENSION);
    int block_num_1d = ceil((float)OUT_DIMENSION / thread_num_1d);
    out_backprop<<<block_num_1d, thread_num_1d>>>(neuron[output_layer_idx], grad_neuron[output_layer_idx], OUT_DIMENSION, y);

    // hidden layers
    for (int i = output_layer_idx-1; i >= 0; i--) {
        int width = (i == 0 ? IN_DIMENSION : hidden_size);
        int height = (i == output_layer_idx-1 ? OUT_DIMENSION : hidden_size);

        int block_num = ceil(sqrt((float)thread_num));
        dim3 dim_block(block_num, block_num);
        dim3 dim_grid(ceil((float)height/block_num), ceil((float)width/block_num));
        thread_num_1d = (height > thread_num ? thread_num : height);
        block_num_1d = ceil((float)height / thread_num_1d);

        neuron_backprop<<<dim_grid, dim_block>>>(neuron[i], grad_neuron[i+1], grad_neuron[i], weight[i], width, height);
        relu_backprop<<<block_num_1d, thread_num_1d>>>(neuron[i], grad_neuron[i], height);
        weight_backprop<<<dim_grid, dim_block>>>(neuron[i], grad_neuron[i+1], grad_weight[i], width, height);
        bias_backprop<<<block_num_1d, thread_num_1d>>>(grad_neuron[i+1], grad_bias[i], height);
    }
}

void clear_grad(float** grad_weight, float** grad_bias, int hidden_size, int num_layers) {
    assert(hipMemset(grad_weight[0], 0, IN_DIMENSION * hidden_size * sizeof(float)) == hipSuccess);
    assert(hipMemset(grad_bias[0], 0, hidden_size * sizeof(float)) == hipSuccess);

    for (int i = 1; i < num_layers-2; ++i) {
        assert(hipMemset(grad_weight[i], 0, hidden_size * hidden_size * sizeof(float)) == hipSuccess);
        assert(hipMemset(grad_bias[i], 0, hidden_size * sizeof(float)) == hipSuccess);
    }

    assert(hipMemset(grad_weight[num_layers-2], 0, hidden_size * OUT_DIMENSION * sizeof(float)) == hipSuccess);
    assert(hipMemset(grad_bias[num_layers-2], 0, OUT_DIMENSION * sizeof(float)) == hipSuccess);
}

__global__ void  update_weights(float* w, float* grad_w, int size, float factor) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        w[i] -= factor * grad_w[i];
    }
}

void clear_neurons(float** neurons, int num_layers, int hidden_size) {
    for (int i = 0; i < num_layers; ++i) {
        int size = (i == 0 ? IN_DIMENSION : (i == num_layers-1 ? OUT_DIMENSION : hidden_size));

        assert(hipMemset(neurons[i], 0, size * sizeof(float)) == hipSuccess);
    }
}

void train(float** train_data, int* train_label, int train_size, float** weight, float** bias,
            int hidden_size, int num_layers, float learning_rate, int epochs,
            int batch_size, float* loss, float* accuracy, int thread_num) {
    float** weight_cu = NULL;
    float** bias_cu = NULL;
    float** train_data_cu = NULL;

    assert(hipHostMalloc((void**)&weight_cu, (num_layers-1) * sizeof(float*)) == hipSuccess);
    assert(hipHostMalloc((void**)&bias_cu, (num_layers-1) * sizeof(float*)) == hipSuccess);
    assert(hipHostMalloc((void**)&train_data_cu, train_size * sizeof(float*)) == hipSuccess);

    float** neurons = NULL;
    float** grad_weight = NULL;
    float** grad_bias = NULL;
    float** grad_neuron = NULL;

    assert(hipHostMalloc((void**)&neurons, num_layers * sizeof(float*)) == hipSuccess);
    assert(hipHostMalloc((void**)&grad_weight, (num_layers-1) * sizeof(float*)) == hipSuccess);
    assert(hipHostMalloc((void**)&grad_bias, (num_layers-1) * sizeof(float*)) == hipSuccess);
    assert(hipHostMalloc((void**)&grad_neuron, num_layers * sizeof(float*)) == hipSuccess);

    allocate_memory(neurons, grad_weight, grad_bias, grad_neuron, weight_cu, bias_cu, train_data_cu, hidden_size, num_layers, train_size);

    for (int i = 0; i < train_size; ++i) {
        assert(hipMemcpy(train_data_cu[i], train_data[i], IMAGE_SIZE * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    }

    assert(hipMemcpy(weight_cu[0], weight[0], IN_DIMENSION * hidden_size * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(bias_cu[0], bias[0], hidden_size * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    for (int i = 1; i < num_layers-2; ++i) {
        assert(hipMemcpy(weight_cu[i], weight[i], hidden_size * hidden_size * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
        assert(hipMemcpy(bias_cu[i], bias[i], hidden_size * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    }
    assert(hipMemcpy(weight_cu[num_layers-2], weight[num_layers-2], hidden_size * OUT_DIMENSION * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(bias_cu[num_layers-2], bias[num_layers-2], OUT_DIMENSION * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);

    for (int e = 0; e < epochs; ++e) {
        float loss_sum = 0;
        int acc_cnt = 0;
        for (int j = 0; j < train_size; j += batch_size) {
            clear_grad(grad_weight, grad_bias, hidden_size, num_layers);
            for (int i = j; i < j + batch_size; i++) {
                clear_neurons(neurons, num_layers, hidden_size);
                clear_neurons(grad_neuron, num_layers, hidden_size);
                float loss;
                int idx = i % train_size;
                int y = forward(train_data_cu[idx], IN_DIMENSION, train_label[idx], weight_cu, bias_cu, neurons, hidden_size, num_layers, &loss, thread_num);
                acc_cnt += (y == train_label[idx]);
                backward(train_label[idx], weight_cu, bias_cu, neurons, grad_weight, grad_bias, grad_neuron, hidden_size, num_layers, thread_num);
                loss_sum += loss;

                float factor = learning_rate / batch_size;

                for (int i = 0; i < num_layers-1; ++i) {
                    int width = (i == 0 ? IN_DIMENSION : hidden_size);
                    int height = (i == num_layers-2 ? OUT_DIMENSION : hidden_size);

                    int size = width * height;
                    int thread_num_1d = (size > thread_num ? thread_num : size);
                    int block_num_1d = ceil((float)size / thread_num_1d);
                    update_weights<<<block_num_1d, thread_num_1d>>>(weight_cu[i], grad_weight[i], size, factor);

                    thread_num_1d = (height > thread_num ? thread_num : height);
                    block_num_1d = ceil((float)height / thread_num_1d);
                    update_weights<<<block_num_1d, thread_num_1d>>>(bias_cu[i], grad_bias[i], height, factor);
                }
            }
        }
        *loss = loss_sum / train_size;
        *accuracy = (float)acc_cnt / train_size;
        printf("epochs: %d, loss: %.2f, accuracy: %.2f%%\n", e, *loss, (*accuracy)*100);
    }

    for (int i = 0; i < num_layers-1; ++i) {
        int width = (i == 0 ? IN_DIMENSION : hidden_size);
        int height = (i == num_layers-2 ? OUT_DIMENSION : hidden_size);

        int size = width * height;
        assert(hipMemcpy(weight[i], weight_cu[i], size * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess);

        size = height;
        assert(hipMemcpy(bias[i], bias_cu[i], size * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess);
    }

    hipDeviceSynchronize();

    free_memory(neurons, grad_weight, grad_bias, grad_neuron, weight_cu, bias_cu, train_data_cu, num_layers, train_size);
}

void eval(float** test_data, int* test_label, int test_size, float** weight, float** bias,
            int hidden_size, int num_layers, float* loss, float* accuracy, int thread_num) {
    float** neurons = NULL;
    assert(hipHostMalloc(&neurons, num_layers * sizeof(float*)) == hipSuccess);
    assert(hipMalloc(&neurons[0], IN_DIMENSION * sizeof(float)) == hipSuccess);
    for (int i = 1; i < num_layers-1; ++i) {
        assert(hipMalloc(&neurons[i], hidden_size * sizeof(float)) == hipSuccess);
    }
    assert(hipMalloc(&neurons[num_layers-1], OUT_DIMENSION * sizeof(float)) == hipSuccess);

    float** weight_cu = NULL;
    float** bias_cu = NULL;
    float** test_data_cu = NULL;

    assert(hipHostMalloc((void**)&weight_cu, (num_layers-1) * sizeof(float*)) == hipSuccess);
    assert(hipHostMalloc((void**)&bias_cu, (num_layers-1) * sizeof(float*)) == hipSuccess);
    assert(hipHostMalloc((void**)&test_data_cu, test_size * sizeof(float*)) == hipSuccess);

    allocate_weight_and_data(weight_cu, bias_cu, test_data_cu, hidden_size, num_layers, test_size);

    for (int i = 0; i < test_size; ++i) {
        assert(hipMemcpy(test_data_cu[i], test_data[i], IMAGE_SIZE * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    }

    assert(hipMemcpy(weight_cu[0], weight[0], IN_DIMENSION * hidden_size * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(bias_cu[0], bias[0], hidden_size * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    for (int i = 1; i < num_layers-2; ++i) {
        assert(hipMemcpy(weight_cu[i], weight[i], hidden_size * hidden_size * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
        assert(hipMemcpy(bias_cu[i], bias[i], hidden_size * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    }
    assert(hipMemcpy(weight_cu[num_layers-2], weight[num_layers-2], hidden_size * OUT_DIMENSION * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(bias_cu[num_layers-2], bias[num_layers-2], OUT_DIMENSION * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);

    int cnt = 0;
    float loss_sum = 0;

    for (int i = 0; i < test_size; ++i) {
        clear_neurons(neurons, num_layers, hidden_size);
        int y = forward(test_data_cu[i], IN_DIMENSION, test_label[i], weight_cu, bias_cu, neurons, hidden_size, num_layers, &loss_sum, thread_num);
        cnt += (y == test_label[i]);
    }

    *loss = loss_sum / test_size;
    *accuracy = (float) cnt / test_size;

    for (int i = 0; i < num_layers; ++i) {
        assert(hipFree(neurons[i]) == hipSuccess);
    }
    assert(hipHostFree(neurons) == hipSuccess);
    free_weight_and_data(weight_cu, bias_cu, test_data_cu, num_layers, test_size);
}

int main(int argc, char** argv) {
    if (argc != 7) {
        printf("Usage: %s <layers> <units> <epochs> <batch size> <learning rate> <thread_num>\n", argv[0]);
        return 1;
    }
    int nl = atoi(argv[1]) + 2;
    int nh = atoi(argv[2]);
    int ne = atoi(argv[3]);
    int nb = atoi(argv[4]);
    float lr = atof(argv[5]);
    int weights_num = nl - 1;

    int thread_num = atoi(argv[6]);

    srand(time(NULL));

    float** train_set = NULL;
    int* train_labels = NULL;
    assert(hipHostMalloc((void**) &train_set, sizeof(float*) * TRAINING_SET_SIZE) == hipSuccess);
    assert(hipHostMalloc((void**) &train_labels, sizeof(int) * TRAINING_SET_SIZE) == hipSuccess);
    read_data_set(train_set, TRAIN_IMAGE_FILE, train_labels, TRAIN_LABEL_FILE, TRAINING_SET_SIZE);


    float** test_set = NULL;
    int* test_labels = NULL;
    assert(hipHostMalloc((void**) &test_set, sizeof(float*) * TEST_SET_SIZE) == hipSuccess);
    assert(hipHostMalloc((void**) &test_labels, sizeof(int) * TEST_SET_SIZE) == hipSuccess);
    read_data_set(test_set, TEST_IMAGE_FILE, test_labels, TEST_LABEL_FILE, TEST_SET_SIZE);

    float** weight = NULL;
    assert(hipHostMalloc((void**) &weight, sizeof(float*) * weights_num) == hipSuccess);
    assert(hipHostMalloc((void**) &weight[0], sizeof(float) * IMAGE_SIZE * nh) == hipSuccess);
    kaiming_init(weight[0], nh, IMAGE_SIZE);
    for (int i = 1; i < weights_num-1; i++) {
        assert(hipHostMalloc((void**) &weight[i], sizeof(float) * nh * nh) == hipSuccess);
        kaiming_init(weight[i], nh, nh);
    }
    assert(hipHostMalloc((void**) &weight[weights_num-1], sizeof(float) * nh * OUT_DIMENSION) == hipSuccess);
    kaiming_init(weight[weights_num-1], OUT_DIMENSION, nh);

    float** bias = NULL;
    assert(hipHostMalloc((void**) &bias, sizeof(float*) * weights_num) == hipSuccess);
    for (int i = 0; i < weights_num-1; ++i) {
        assert(hipHostMalloc((void**) &bias[i], sizeof(float) * nh) == hipSuccess);
        kaiming_init(bias[i], nh, 1);
    }
    assert(hipHostMalloc((void**) &bias[weights_num-1], sizeof(float) * OUT_DIMENSION) == hipSuccess);
    kaiming_init(bias[weights_num-1], OUT_DIMENSION, 1);

    float loss = 0;
    float accuracy = 0;

    // Measure running time for training
    clock_t start, end;
    start = clock();
    train(train_set, train_labels, TRAINING_SET_SIZE, weight, bias, nh, nl, lr, ne, nb, &loss, &accuracy, thread_num);
    end = clock();

    float time = ((float) (end - start)) / CLOCKS_PER_SEC;
    printf("Time for training: %.2lfs, grind rate: %.2lf, loss: %.2f, accuracy: %.2f%%\n", time, ((float)TRAINING_SET_SIZE*ne)/time, loss, accuracy*100);

    loss = 0;
    accuracy = 0;
    // Measure running time for testing
    start = clock();
    eval(test_set, test_labels, TEST_SET_SIZE, weight, bias, nh, nl, &loss, &accuracy, thread_num);
    end = clock();

    time = ((float) (end - start)) / CLOCKS_PER_SEC;
    printf("Time for validation: %.2lfs, grind rate: %.2lf, loss: %.2f, accuracy: %.2f%%\n", time, (float)TEST_SET_SIZE/time, loss, accuracy*100);

    free_data_set(train_set, train_labels, TRAINING_SET_SIZE);
    free_data_set(test_set, test_labels, TEST_SET_SIZE);
    for (int i = 0; i < weights_num; ++i) {
        assert(hipHostFree(weight[i]) == hipSuccess);
        assert(hipHostFree(bias[i]) == hipSuccess);
    }
    assert(hipHostFree(weight) == hipSuccess);
    assert(hipHostFree(bias) == hipSuccess);
}